
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * Refactor the `helloGPU` definition to be a kernel
 * that can be launched on the GPU. Update its message
 * to read "Hello from the GPU!"
 */

__global__  void helloGPU()
{
  printf("Hello from the GPU.\n");
}

int main()
{


  /*
   * Refactor this call to `helloGPU` so that it launches
   * as a kernel on the GPU.
   */

  helloGPU<<<1,2>>>();
  helloCPU<<<1,1>>>();
  helloGPU<<<1,1>>>();

  /*
   * Add code below to synchronize on the completion of the
   * `helloGPU` kernel completion before continuing the CPU
   * thread.
   */
   hipDeviceSynchronize();
}
