
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N; i += stride)
  {
  result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  checkCuda( hipMallocManaged(&a,size));
  checkCuda( hipMallocManaged(&b,size));
  checkCuda( hipMallocManaged(&c,size));
  
  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  size_t threads_per_block = 1024;
  size_t number_of_blocks = ( N+ threads_per_block-1)/threads_per_block;
  
  addVectorsInto<<<number_of_blocks, threads_per_block>>>(c, a, b, N);

  checkCuda( hipGetLastError());
  checkCuda( hipDeviceSynchronize());

  checkElementsAre(7, c, N);
  checkCuda( hipFree(a));
  checkCuda( hipFree(b));
  checkCuda( hipFree(c));
}
